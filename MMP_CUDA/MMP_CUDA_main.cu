#include "hip/hip_runtime.h"
//��� �� CUDA
#include <iostream>
#include <fstream>
#include <string>
#include <ctime>
#include <iomanip>

#include <windows.h>

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#define CUDA_CALL( call )               \
{                                       \
hipError_t result = call;              \
if ( hipSuccess != result )            \
    std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

using namespace std;

// Constants

const int S = 134862;
const int m = 247;
const int K = S / m;

const int grid_size_m = 13;
const int block_size_m = 19;

const int per_thread_m = m / (grid_size_m * block_size_m);

// Submatrices in main algorithm

double a[K][m*m];
double b[K][m*m];
double c[K][m*m];
double d[K][m*m];
double e[K][m*m];
double f[K][m];

// Temporal variables in main algorithm

double tmpv[m];
double tmpv2[m];
double tmpv3[m];
double tmpm[m*m];
double tmpm2[m*m];
double tmpm3[m*m];
double delta[m*m];
double alpha[K - 1][m*m];
double beta[K - 2][m*m];
double gamma[K][m];

// Solution of SLAE
double y[K][m];

// Device variables

double *dev_a = new double[m*m];
double *dev_b = new double[m*m];
double *dev_c = new double[m*m];
double *dev_d = new double[m*m];
double *dev_e = new double[m*m];
double *dev_f = new double[m];

double *dev_tmpv = new double[m];
double *dev_tmpv2 = new double[m];
double *dev_tmpv3 = new double[m];
double *dev_tmpm = new double[m*m];
double *dev_tmpm2 = new double[m*m];
double *dev_tmpm3 = new double[m*m];
double *dev_delta = new double[m*m];
double *dev_alpha = new double[m*m];
double *dev_beta = new double[m*m];
double *dev_gamma = new double[m];

double *dev_y = new double[m];

void CUDA_prep() {
	CUDA_CALL(hipMalloc((void**)&dev_a, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_b, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_c, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_d, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_e, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_f, m * sizeof(double)));

	CUDA_CALL(hipMalloc((void**)&dev_tmpv, m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_tmpv2, m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_tmpv3, m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_tmpm, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_tmpm2, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_tmpm3, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_delta, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_alpha, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_beta, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc((void**)&dev_gamma, m * sizeof(double)));

	CUDA_CALL(hipMalloc((void**)&dev_y, m * sizeof(double)));
}

void CUDA_end() {
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(dev_e);
	hipFree(dev_f);

	hipFree(dev_tmpv);
	hipFree(dev_tmpv2);
	hipFree(dev_tmpv3);
	hipFree(dev_tmpm);
	hipFree(dev_tmpm2);
	hipFree(dev_tmpm3);
	hipFree(dev_delta);
	hipFree(dev_alpha);
	hipFree(dev_beta);
	hipFree(dev_gamma);

	hipFree(dev_y);
}


// Functions
void showv(double * ptr, int start, int size) {
	for (int i = start; i < start + size; i++) {
		cout << ptr[i] << endl;
	}
}

void write_in_file(double *X) {
	fstream result_file;
	result_file.open("X_cuda.dat");

	for (int i = 0; i < S; ++i) {
		result_file << X[i] << endl;
	}

	result_file.close();
	cout << endl;
	cout << "Answer was written in file" << endl;
}

//----------------------------------------------------

void addm(double * A, double * B, double * C) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = A[i*m + j] + B[i*m + j];
		}
	}
}

__global__ void GPU_addm(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = A[i*m + j] + B[i*m + j];
		}
	}
}

//----------------------------------------------------


void subm(double * A, double * B, double * C) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = A[i*m + j] - B[i*m + j];
		}
	}
}

__global__ void GPU_subm(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = A[i*m + j] - B[i*m + j];
		}
	}
}

//----------------------------------------------------

void mulm(double * A, double * B, double * C) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = 0;
			for (int k = 0; k < m; k++) {
				C[i*m + j] += A[i*m + k] * B[k*m + j];
			}
		}
	}
}

__global__ void GPU_mulm(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		for (int j = 0; j < m; j++) {
			C[i*m + j] = 0;
			for (int k = 0; k < m; k++) {
				C[i*m + j] += A[i*m + k] * B[k*m + j];
			}
		}
	}
}

//----------------------------------------------------

void addv(double * A, double * B, double * C) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		C[i] = A[i] + B[i];
	}
}

__global__ void GPU_addv(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		C[i] = A[i] + B[i];
	}
}

//----------------------------------------------------

void subv(double * A, double * B, double * C) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		C[i] = A[i] - B[i];
	}
}

__global__ void GPU_subv(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		C[i] = A[i] - B[i];
	}
}

//----------------------------------------------------

void mulmv(double * A, double * B, double * C) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		C[i] = 0;
		for (int j = 0; j < m; j++) {
			C[i] += A[i*m + j] * B[j];
		}
	}
}


__global__ void GPU_mulmv(double * A, double * B, double * C) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		C[i] = 0;
		for (int j = 0; j < m; j++) {
			C[i] += A[i*m + j] * B[j];
		}
	}
}

//----------------------------------------------------

void copyv(double * A, double * B) {
#pragma omp parallel for
	for (int i = 0; i < m; i++) {
		B[i] = A[i];
	}
}

__global__ void GPU_copyv(double * A, double * B) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * per_thread_m; i < (thread + 1) * per_thread_m; i++) {
		B[i] = A[i];
	}
}

//----------------------------------------------------

void copym(double * A, double * B) {
#pragma omp parallel for
	for (int i = 0; i < m*m; i++) {
		B[i] = A[i];
	}
}

//<<<13, 19>>>
__global__ void GPU_copym(double * A, double * B) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = thread * m; i < (thread + 1) * m; i++) {
		B[i] = A[i];
	}
}

//----------------------------------------------------

//��� ������ = size - thread;

void solvev(double * A, double * B, double * X) {
	// Gaussian elimination 
	double tmpA[m*m];
	copym(A, tmpA);
	copyv(B, X);

	// Direct
	for (int d = 0; d < m; d++) {
		// dividing on diagonal element
		if (tmpA[d*m + d] != 1) {
			X[d] /= tmpA[d*m + d];
			for (int j = m - 1; j > d; j--) {
				tmpA[d*m + j] /= tmpA[d*m + d];
			}
			tmpA[d*m + d] = 1;
		}

		// nulling elements below diagonal in column
		for (int i = d + 1; i < m; i++) {
			if (tmpA[i*m + d] != 0) {
				double mult = -tmpA[i*m + d] / tmpA[d*m + d];
				X[i] += mult*X[d];
				for (int j = d; j < m; j++) {
					tmpA[i*m + j] += mult*tmpA[d*m + j];
				}
			}
		}
	}

	// Inverse
	for (int d = m - 1; d >= 0; d--) {
		// nulling elements above diagonal in column
		for (int i = d - 1; i >= 0; i--) {
			if (tmpA[i*m + d] != 0) {
				double mult = -tmpA[i*m + d] / tmpA[d*m + d];
				X[i] += mult*X[d];
				for (int j = d; j < m; j++) {
					tmpA[i*m + j] += mult*tmpA[d*m + j];
				}
			}
		}
	}
}

//__global__ void GPU_solvev(double * A, double * B, double * X) {
//	// Gaussian elimination 
//	int thread = blockIdx.x * blockDim.x + threadIdx.x;
//	
//	double tmpA[m*m];
//	double mult = 0;
//	copym(A, tmpA);
//	copyv(B, X);
//
//	// Direct
//	for (int d = thread * per_thread_m; d < (thread + 1) * per_thread_m; d++) {
//		// dividing on diagonal element
//		if (tmpA[d*m + d] != 1) {
//			X[d] /= tmpA[d*m + d];
//			for (int j = m - 1; j > d; j--) {
//				tmpA[d*m + j] /= tmpA[d*m + d];
//			}
//			tmpA[d*m + d] = 1;
//		}
//
//		// nulling elements below diagonal in column
//		for (int i = d + 1; i < m; i++) {
//			if (tmpA[i*m + d] != 0) {
//				mult = -tmpA[i*m + d] / tmpA[d*m + d];
//				X[i] += mult*X[d];
//				for (int j = d; j < m; j++) {
//					tmpA[i*m + j] += mult*tmpA[d*m + j];
//				}
//			}
//		}
//	}
//
//	// Inverse
//	for (int d = ((thread + 1) * per_thread_m) - 1; d >= thread * per_thread_m; --d) {
//	//for (int d = m - 1; d >= 0; d--) {
//		// nulling elements above diagonal in column
//		for (int i = d - 1; i >= 0; i--) {
//			if (tmpA[i*m + d] != 0) {
//				mult = -tmpA[i*m + d] / tmpA[d*m + d];
//				X[i] += mult*X[d];
//				for (int j = d; j < m; j++) {
//					tmpA[i*m + j] += mult*tmpA[d*m + j];
//				}
//			}
//		}
//	}
//}

//----------------------------------------------------

void solvem(double * A, double * B, double * X) {
	// Gaussian elimination 
	double tmpA[m*m];
	copym(B, X);
	copym(A, tmpA);

	// Direct

	for (int d = 0; d < m; d++) {
		if (tmpA[d*m + d] != 1) {
			for (int j = 0; j < m; j++) {
				X[d*m + j] /= tmpA[d*m + d];
			}
			for (int j = m - 1; j >= 0; j--) {
				tmpA[d*m + j] /= tmpA[d*m + d];
			}
		}

		for (int i = d + 1; i < m; i++) {
			if (tmpA[i*m + d] != 0) {
				double mult = -tmpA[i*m + d] / tmpA[d*m + d];
				for (int j = 0; j < m; j++) {
					X[i*m + j] += mult*X[d*m + j];
				}
				for (int j = 0; j < m; j++) {
					tmpA[i*m + j] += mult*tmpA[d*m + j];
				}
			}
		}
	}

	// Inverse

	for (int d = m - 1; d >= 0; d--) {
		for (int i = d - 1; i >= 0; i--) {
			if (tmpA[i*m + d] != 0) {
				double mult = -tmpA[i*m + d] / tmpA[d*m + d];
				for (int j = 0; j < m; j++) {
					X[i*m + j] += mult*X[d*m + j];
				}
				for (int j = d; j < m; j++) {
					tmpA[i*m + j] += mult*tmpA[d*m + j];
				}
			}
		}
	}
}

//__global__ void GPU_solvem(double * A, double * B, double * X) {
//	// Gaussian elimination 
//	int thread = blockIdx.x * blockDim.x + threadIdx.x;
//	double mult = 0;
//	double tmpA[m*m];
//	copym(B, X);
//	copym(A, tmpA);
//
//	// Direct
//
//	for (int d = thread * per_thread_m; d < (thread + 1) * per_thread_m; d++) {
//		if (tmpA[d*m + d] != 1) {
//			for (int j = 0; j < m; j++) {
//				X[d*m + j] /= tmpA[d*m + d];
//			}
//			for (int j = m - 1; j >= 0; j--) {
//				tmpA[d*m + j] /= tmpA[d*m + d];
//			}
//		}
//
//		for (int i = d + 1; i < m; i++) {
//			if (tmpA[i*m + d] != 0) {
//				mult = -tmpA[i*m + d] / tmpA[d*m + d];
//				for (int j = 0; j < m; j++) {
//					X[i*m + j] += mult*X[d*m + j];
//				}
//				for (int j = 0; j < m; j++) {
//					tmpA[i*m + j] += mult*tmpA[d*m + j];
//				}
//			}
//		}
//	}
//
//	// Inverse
//
//	for (int d = ((thread + 1) * per_thread_m) - 1; d >= thread * per_thread_m; --d) {
//		for (int i = d - 1; i >= 0; i--) {
//			if (tmpA[i*m + d] != 0) {
//				mult = -tmpA[i*m + d] / tmpA[d*m + d];
//				for (int j = 0; j < m; j++) {
//					X[i*m + j] += mult*X[d*m + j];
//				}
//				for (int j = d; j < m; j++) {
//					tmpA[i*m + j] += mult*tmpA[d*m + j];
//				}
//			}
//		}
//	}
//}

//----------------------------------------------------

// Preparations
int prep() {
	string tmps;
	double tmpd;
	ifstream A1f;
	ifstream A2f;
	ifstream A3f;
	ifstream A4f;
	ifstream A5f;
	ifstream Ff;

	A1f.open("A1.dat");
	A2f.open("A2.dat");
	A3f.open("A3.dat");
	A4f.open("A4.dat");
	A5f.open("A5.dat");
	Ff.open("F");

	if (A1f.is_open() && A2f.is_open() && A3f.is_open()
		&& A4f.is_open() && A5f.is_open() && Ff.is_open()) {
		std::getline(A1f, tmps);
		std::getline(A2f, tmps);
		std::getline(A3f, tmps);
		std::getline(A4f, tmps);
		std::getline(A5f, tmps);
		std::getline(Ff, tmps);

		A2f >> tmpd;
		for (int i = 0; i < m; i++) {
			A1f >> tmpd;
		}

		for (int i = 0; i < K; i++) {
			for (int j = 0; j < m; j++) {
				A3f >> c[i][j*m + j];
				if (j - 1 >= 0) {
					A2f >> c[i][j*m + j - 1];
				}
				if (j + 1 < m) {
					A4f >> c[i][j*m + j + 1];
				}
				if (i >= 1) {
					A1f >> b[i][j*m + j];
					b[i][j*m + j] *= -1;
				}
				if (i < K - 1) {
					A5f >> d[i][j*m + j];
					d[i][j*m + j] *= -1;
				}
				if (i >= 2) {
					a[i][j*m + j] = 0.001;
				}
				if (i < K - 2) {
					e[i][j*m + j] = 0.001;
				}
				Ff >> f[i][j];
			}
			A2f >> tmpd;
			A4f >> tmpd;
		}

		A1f.close();
		A2f.close();
		A3f.close();
		A4f.close();
		A5f.close();
		Ff.close();
	}
	else {
		std::cout << "File error, program aborted" << std::endl;
		return 1;
	}

	return 0;
}

// Main algorithm
int algo() {

	CUDA_CALL(hipMalloc(&dev_a, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_b, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_c, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_d, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_e, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_f, m * sizeof(double)));

	CUDA_CALL(hipMalloc(&dev_tmpv, m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_tmpv2, m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_tmpv3, m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_tmpm, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_tmpm2, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_tmpm3, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_delta, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_alpha, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_beta, m * m * sizeof(double)));
	CUDA_CALL(hipMalloc(&dev_gamma, m * sizeof(double)));

	CUDA_CALL(hipMalloc(&dev_y, m * sizeof(double)));

	//----------------------------------------------------

	cout << "first start" << endl;
	solvem(c[0], d[0], alpha[0]);
	solvem(c[0], e[0], beta[0]);
	solvev(c[0], f[0], gamma[0]);

	//----------------------------------------------------

	cout << "second start" << endl;
	CUDA_CALL(hipMemcpy(dev_b, b[1], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_alpha, alpha[0], m * m * sizeof(double), hipMemcpyHostToDevice));
	//CUDA_CALL(hipMemcpy(dev_tmpm, tmpm, m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_c, c[1], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(b[1], alpha[0], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_b, dev_alpha, dev_tmpm);
	//subm(c[1], tmpm, delta);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_c, dev_tmpm, dev_delta);

	//----------------------------------------------------

	cout << "third start" << endl;
	CUDA_CALL(hipMemcpy(dev_beta, beta[0], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_d, d[1], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(b[1], beta[0], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_b, dev_beta, dev_tmpm);
	//subm(d[1], tmpm, tmpm2);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_d, dev_tmpm, dev_tmpm2);

	CUDA_CALL(hipMemcpy(delta, dev_delta, m * m * sizeof(double), hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(tmpm2, dev_tmpm2, m * m * sizeof(double), hipMemcpyDeviceToHost));

	solvem(delta, tmpm2, alpha[1]);
	solvem(delta, e[1], beta[1]);

	CUDA_CALL(hipMemcpy(dev_b, b[1], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_gamma, gamma[0], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_f, f[1], m * sizeof(double), hipMemcpyHostToDevice));

	//mulmv(b[1], gamma[0], tmpv);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_b, dev_gamma, dev_tmpv);
	//addv(f[1], tmpv, tmpv2);
	GPU_addv<<<grid_size_m, block_size_m>>>(dev_f, dev_tmpv, dev_tmpv2);

	CUDA_CALL(hipMemcpy(tmpv2, dev_tmpv2, m * sizeof(double), hipMemcpyDeviceToHost));

	solvev(delta, tmpv2, gamma[1]);

	//----------------------------------------------------

	cout << "cycle start" << endl;
	for (int i = 2; i < K - 2; i++) {
		cout << "i = " << i << " / " << K - 2 << endl;
		
		CUDA_CALL(hipMemcpy(dev_a, a[i], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_alpha, alpha[i - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_b, b[i], m * m * sizeof(double), hipMemcpyHostToDevice));

		//mulm(a[i], alpha[i - 2], tmpm);
		GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_alpha, dev_tmpm);
		//subm(tmpm, b[i], tmpm3);
		GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_b, dev_tmpm3);

		//----------------------------------------------------

		CUDA_CALL(hipMemcpy(dev_alpha, alpha[i - 1], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_c, c[i], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_beta, beta[i - 2], m * m * sizeof(double), hipMemcpyHostToDevice));

		//mulm(tmpm3, alpha[i - 1], tmpm);
		GPU_mulm<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_alpha, dev_tmpm);

		//addm(tmpm, c[i], tmpm2);
		GPU_addm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_c, dev_tmpm2);

		//mulm(a[i], beta[i - 2], tmpm);
		GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_beta, dev_tmpm);

		//subm(tmpm2, tmpm, delta);
		GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_delta);

		//----------------------------------------------------

		CUDA_CALL(hipMemcpy(dev_beta, beta[i - 1], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_d, d[i], m * m * sizeof(double), hipMemcpyHostToDevice));

		//mulm(tmpm3, beta[i - 1], tmpm);
		GPU_mulm<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_beta, dev_tmpm);

		//addm(tmpm, d[i], tmpm2);
		GPU_addm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_d, dev_tmpm2);

		CUDA_CALL(hipMemcpy(delta, dev_delta, m * m * sizeof(double), hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(tmpm2, dev_tmpm2, m * m * sizeof(double), hipMemcpyDeviceToHost));

		solvem(delta, tmpm2, alpha[i]);

		//----------------------------------------------------
		
		solvem(delta, e[i], beta[i]);

		//----------------------------------------------------

		CUDA_CALL(hipMemcpy(dev_gamma, gamma[i - 1], m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_f, f[i], m * sizeof(double), hipMemcpyHostToDevice));

		//mulmv(tmpm3, gamma[i - 1], tmpm);
		GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_gamma, dev_tmpm);

		//subv(f[i], tmpm, tmpm2);
		GPU_subv<<<grid_size_m, block_size_m>>>(dev_f, dev_tmpm, dev_tmpm2);

		CUDA_CALL(hipMemcpy(dev_gamma, gamma[i - 2], m * sizeof(double), hipMemcpyHostToDevice));

		//mulmv(a[i], gamma[i - 2], tmpm);
		GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_a, dev_gamma, dev_tmpm);

		//subv(tmpm2, tmpm, tmpm3);
		GPU_subv<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_tmpm3);

		CUDA_CALL(hipMemcpy(tmpm3, dev_tmpm3, m * m * sizeof(double), hipMemcpyDeviceToHost));

		solvev(delta, tmpm3, gamma[i]);
	}

	//----------------------------------------------------

	cout << "fourth start" << endl;

	CUDA_CALL(hipMemcpy(dev_a, a[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_alpha, alpha[K - 4], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_b, b[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
	
	//mulm(a[K - 2], alpha[K - 4], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_alpha, dev_tmpm);

	//subm(tmpm, b[K - 2], tmpm3);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_b, dev_tmpm3);

	CUDA_CALL(hipMemcpy(dev_a, a[K - 3], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_c, c[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
	
	//mulm(tmpm3, alpha[K - 3], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_alpha, dev_tmpm);
	
	//addm(tmpm, c[K - 2], tmpm2);
	GPU_addm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_c, dev_tmpm2);

	CUDA_CALL(hipMemcpy(dev_a, a[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_beta, beta[K - 4], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(a[K - 2], beta[K - 4], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_beta, dev_tmpm);

	//subm(tmpm2, tmpm, delta);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_delta);
	
	//----------------------------------------------------

	cout << "fifth start" << endl;

	CUDA_CALL(hipMemcpy(dev_beta, beta[K - 3], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_d, d[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(tmpm3, beta[K - 3], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_beta, dev_tmpm);

	//addm(tmpm, d[K - 2], tmpm2);
	GPU_addm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_d, dev_tmpm2);

	CUDA_CALL(hipMemcpy(delta, dev_delta, m * m * sizeof(double), hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(tmpm2, dev_tmpm2, m * m * sizeof(double), hipMemcpyDeviceToHost));

	solvem(delta, tmpm2, alpha[K - 2]);

	//----------------------------------------------------

	cout << "sixth start" << endl;

	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 3], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_f, f[K - 2], m * sizeof(double), hipMemcpyHostToDevice));

	//mulmv(tmpm3, gamma[K - 3], tmpm);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_gamma, dev_tmpm);

	//subv(f[K - 2], tmpm, tmpm2);
	GPU_subv<<<grid_size_m, block_size_m>>>(dev_f, dev_tmpm, dev_tmpm2);

	CUDA_CALL(hipMemcpy(dev_a, a[K - 2], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 4], m * sizeof(double), hipMemcpyHostToDevice));


	//mulmv(a[K - 2], gamma[K - 4], tmpm);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_a, dev_gamma, dev_tmpm);

	//subv(tmpm2, tmpm, tmpm3);
	GPU_subv<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_tmpm3);

	CUDA_CALL(hipMemcpy(tmpm3, dev_tmpm3, m * m * sizeof(double), hipMemcpyDeviceToHost));

	solvev(delta, tmpm3, gamma[K - 2]);

	//----------------------------------------------------

	cout << "seventh start" << endl;

	CUDA_CALL(hipMemcpy(dev_a, a[K - 1], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_alpha, alpha[K - 3], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_b, b[K - 1], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(a[K - 1], alpha[K - 3], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_alpha, dev_tmpm);

	//subm(tmpm, b[K - 1], tmpm3);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_b, dev_tmpm3);

	CUDA_CALL(hipMemcpy(dev_alpha, alpha[K - 2], m * m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_c, c[K - 1], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(tmpm3, alpha[K - 2], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_alpha, dev_tmpm);

	//addm(tmpm, c[K - 1], tmpm2);
	GPU_addm<<<grid_size_m, block_size_m>>>(dev_tmpm, dev_c, dev_tmpm2);

	CUDA_CALL(hipMemcpy(dev_beta, beta[K - 3], m * m * sizeof(double), hipMemcpyHostToDevice));

	//mulm(a[K - 1], beta[K - 3], tmpm);
	GPU_mulm<<<grid_size_m, block_size_m>>>(dev_a, dev_beta, dev_tmpm);

	//subm(tmpm2, tmpm, delta);
	GPU_subm<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_delta);

	//----------------------------------------------------

	cout << "eight start" << endl;

	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 2], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_f, f[K - 1], m * sizeof(double), hipMemcpyHostToDevice));

	//mulmv(tmpm3, gamma[K - 2], tmpm);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_tmpm3, dev_gamma, dev_tmpm);

	//subv(f[K - 1], tmpm, tmpm2);
	GPU_subv<<<grid_size_m, block_size_m>>>(dev_f, dev_tmpm, dev_tmpm2);

	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 3], m * sizeof(double), hipMemcpyHostToDevice));

	//mulmv(a[K - 1], gamma[K - 3], tmpm);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_a, dev_gamma, dev_tmpm);

	//subv(tmpm2, tmpm, tmpm3);
	GPU_subv<<<grid_size_m, block_size_m>>>(dev_tmpm2, dev_tmpm, dev_tmpm3);

	CUDA_CALL(hipMemcpy(delta, dev_delta, m * m * sizeof(double), hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(tmpm3, dev_tmpm3, m * m * sizeof(double), hipMemcpyDeviceToHost));

	solvev(delta, tmpm3, gamma[K - 1]);

	//----------------------------------------------------

	cout << "nine start" << endl;

	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 1], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_y, y[K - 1], m * sizeof(double), hipMemcpyHostToDevice));

	//copyv(gamma[K - 1], y[K - 1]);
	CUDA_CALL(hipMemcpy(dev_y, dev_gamma, m * sizeof(double), hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpy(y[K - 1], dev_y, m * sizeof(double), hipMemcpyDeviceToHost));

	//mulmv(alpha[K - 2], y[K - 1], tmpv);
	GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_alpha, dev_y, dev_tmpv);

	CUDA_CALL(hipMemcpy(dev_gamma, gamma[K - 2], m * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_y, y[K - 2], m * sizeof(double), hipMemcpyHostToDevice));
	
	//addv(tmpv, gamma[K - 2], y[K - 2]);
	GPU_addv<<<grid_size_m, block_size_m>>>(dev_tmpv, dev_gamma, dev_y);
	CUDA_CALL(hipMemcpy(y[K - 2], dev_y, m * sizeof(double), hipMemcpyDeviceToHost));

	//----------------------------------------------------

	//�������������� ������ ���?

	cout << "second cycle start" << endl;
	for (int i = K - 3; i >= 0; i--) {

		cout << "i = " << i << " / " << K - 3 << endl;

		CUDA_CALL(hipMemcpy(dev_alpha, alpha[i], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_y, y[i + 1], m * sizeof(double), hipMemcpyHostToDevice));

		//mulmv(alpha[i], y[i + 1], tmpv);
		GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_alpha, dev_y, dev_tmpv);

		CUDA_CALL(hipMemcpy(dev_beta, beta[i], m * m * sizeof(double), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_y, y[i + 2], m * sizeof(double), hipMemcpyHostToDevice));

		//mulmv(beta[i], y[i + 2], tmpv2);
		GPU_mulmv<<<grid_size_m, block_size_m>>>(dev_alpha, dev_y, dev_tmpv2);

		//subv(tmpv, tmpv2, tmpv3);
		GPU_subv<<<grid_size_m, block_size_m>>>(dev_tmpv, dev_tmpv2, dev_tmpv3);

		CUDA_CALL(hipMemcpy(dev_gamma, gamma[i], m * sizeof(double), hipMemcpyHostToDevice));

		//addv(tmpv3, gamma[i], y[i]);
		GPU_addv<<<grid_size_m, block_size_m>>>(dev_tmpv3, dev_gamma, dev_y);

		CUDA_CALL(hipMemcpy(y[i], dev_y, m * sizeof(double), hipMemcpyDeviceToHost));

	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(dev_e);
	hipFree(dev_f);

	hipFree(dev_tmpv);
	hipFree(dev_tmpv2);
	hipFree(dev_tmpv3);
	hipFree(dev_tmpm);
	hipFree(dev_tmpm2);
	hipFree(dev_tmpm3);
	hipFree(dev_delta);
	hipFree(dev_alpha);
	hipFree(dev_beta);
	hipFree(dev_gamma);

	hipFree(dev_y);

	return 0;
}

// Entry point
int main() {
	// prepare

	clock_t begin = clock();
	if (prep() != 0) {
		cout << "1 No" << endl;
		return 1;
	}
	clock_t end = clock();
	std::cout << "Preparations = " << double(end - begin)
		/ CLOCKS_PER_SEC << " seconds" << std::endl;

	// CUDA_prep
	//CUDA_prep();

	// algorithm

	CUDA_CALL(hipDeviceSynchronize());

	begin = clock();
	if (algo() != 0) {
		cout << "2 No" << endl;
		return 1;
	}
	end = clock();
	std::cout << "Algorithm = " << double(end - begin)
		/ CLOCKS_PER_SEC << " seconds" << std::endl;

	// CUDA_end
	//CUDA_end();

	// show result and exit

	//showv(y[0], 0, S);
	write_in_file(y[0]);

	system("pause");
	return 0;


	//Runtime: 728.236 sec || 13 x 19
}